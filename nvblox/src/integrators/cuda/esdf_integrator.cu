#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/core/bounding_boxes.h"
#include "nvblox/core/bounding_spheres.h"
#include "nvblox/gpu_hash/cuda/gpu_hash_interface.cuh"
#include "nvblox/gpu_hash/cuda/gpu_indexing.cuh"
#include "nvblox/gpu_hash/cuda/gpu_set.cuh"
#include "nvblox/utils/timing.h"

#include "nvblox/integrators/esdf_integrator.h"

namespace nvblox {

EsdfIntegrator::~EsdfIntegrator() {
  if (cuda_stream_ != nullptr) {
    hipStreamDestroy(cuda_stream_);
  }
}

void EsdfIntegrator::integrateBlocksOnGPU(
    const TsdfLayer& tsdf_layer, const std::vector<Index3D>& block_indices,
    EsdfLayer* esdf_layer) {
  timing::Timer esdf_timer("esdf/integrate");

  if (block_indices.empty()) {
    return;
  }

  // First, check if the stream exists. If not, create one.
  if (cuda_stream_ == nullptr) {
    checkCudaErrors(hipStreamCreate(&cuda_stream_));
  }

  timing::Timer allocate_timer("esdf/integrate/allocate");
  // First, allocate all the destination blocks.
  allocateBlocksOnCPU(block_indices, esdf_layer);
  allocate_timer.Stop();

  timing::Timer mark_timer("esdf/integrate/mark_sites");
  // Then, mark all the sites on GPU.
  // This finds all the blocks that are eligible to be parents.
  markAllSitesCombined(tsdf_layer, block_indices, esdf_layer,
                       &updated_indices_device_, &to_clear_indices_device_);
  mark_timer.Stop();

  if (!to_clear_indices_device_.empty()) {
    timing::Timer compute_timer("esdf/integrate/clear");
    clearAllInvalid(to_clear_indices_device_.toVector(), esdf_layer,
                    &cleared_block_indices_device_);
  }

  timing::Timer compute_timer("esdf/integrate/compute");
  // Parallel block banding on GPU.
  computeEsdfCombined(updated_indices_device_, esdf_layer);
  if (!cleared_block_indices_device_.empty()) {
    computeEsdfCombined(cleared_block_indices_device_, esdf_layer);
  }
  compute_timer.Stop();
}

void EsdfIntegrator::integrateSliceOnGPU(
    const TsdfLayer& tsdf_layer, const std::vector<Index3D>& block_indices,
    float z_min, float z_max, float z_output, EsdfLayer* esdf_layer) {
  timing::Timer esdf_timer("esdf/integrate_slice");

  if (block_indices.empty()) {
    return;
  }

  // First, check if the stream exists. If not, create one.
  if (cuda_stream_ == nullptr) {
    checkCudaErrors(hipStreamCreate(&cuda_stream_));
  }

  timing::Timer mark_timer("esdf/integrate_slice/mark_sites");
  // Then, mark all the sites on GPU.
  // This finds all the blocks that are eligible to be parents.
  markSitesInSliceCombined(tsdf_layer, block_indices, z_min, z_max, z_output,
                           esdf_layer, &updated_indices_device_,
                           &to_clear_indices_device_);
  mark_timer.Stop();

  if (!to_clear_indices_device_.empty()) {
    timing::Timer compute_timer("esdf/integrate/clear");
    clearAllInvalid(to_clear_indices_device_.toVector(), esdf_layer,
                    &cleared_block_indices_device_);
  }

  timing::Timer compute_timer("esdf/integrate_slice/compute");
  // Parallel block banding on GPU.
  computeEsdfCombined(updated_indices_device_, esdf_layer);
  if (!cleared_block_indices_device_.empty()) {
    computeEsdfCombined(cleared_block_indices_device_, esdf_layer);
  }
  compute_timer.Stop();
}

__device__ void clearVoxelDevice(EsdfVoxel* voxel,
                                 float max_squared_distance_vox) {
  voxel->parent_direction.setZero();
  voxel->squared_distance_vox = max_squared_distance_vox;
}

// Mark sites to lower & clear.
// Block size MUST be voxels_per_side x voxels_per_side x voxel_per_size.
// Grid size can be anything.
__global__ void markAllSitesCombinedKernel(
    int num_blocks, Index3D* block_indices,
    Index3DDeviceHashMapType<TsdfBlock> tsdf_block_hash,
    Index3DDeviceHashMapType<EsdfBlock> esdf_block_hash,
    float max_site_distance_m, float min_weight, float max_squared_distance_vox,
    Index3D* updated_vec, int* updated_vec_size, Index3D* to_clear_vec,
    int* to_clear_vec_size) {
  dim3 voxel_index = threadIdx;
  int block_idx = blockIdx.x;

  __shared__ TsdfBlock* tsdf_block;
  __shared__ EsdfBlock* esdf_block;
  __shared__ int updated;
  __shared__ int to_clear;
  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    tsdf_block = nullptr;
    esdf_block = nullptr;
    updated = false;
    to_clear = false;
    auto tsdf_it = tsdf_block_hash.find(block_indices[block_idx]);
    if (tsdf_it != tsdf_block_hash.end()) {
      tsdf_block = tsdf_it->second;
    }
    auto esdf_it = esdf_block_hash.find(block_indices[block_idx]);
    if (esdf_it != esdf_block_hash.end()) {
      esdf_block = esdf_it->second;
    }
  }
  __syncthreads();
  if (tsdf_block == nullptr || esdf_block == nullptr) {
    return;
  }

  // Get the correct voxel for this index.
  const TsdfVoxel* tsdf_voxel =
      &tsdf_block->voxels[voxel_index.x][voxel_index.y][voxel_index.z];
  EsdfVoxel* esdf_voxel =
      &esdf_block->voxels[voxel_index.x][voxel_index.y][voxel_index.z];
  if (tsdf_voxel->weight >= min_weight) {
    // Mark as inside if the voxel distance is negative.
    bool is_inside = tsdf_voxel->distance <= 0.0f;
    if (esdf_voxel->is_inside && is_inside == false) {
      clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
      to_clear = true;
    }
    esdf_voxel->is_inside = is_inside;
    if (is_inside && fabsf(tsdf_voxel->distance) <= max_site_distance_m) {
      esdf_voxel->is_site = true;
      esdf_voxel->squared_distance_vox = 0.0f;
      esdf_voxel->parent_direction.setZero();
      updated = true;
    } else {
      if (esdf_voxel->is_site) {
        esdf_voxel->is_site = false;
        // This voxel needs to be cleared.
        clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
        to_clear = true;
      } else if (!esdf_voxel->observed) {
        // This is a brand new voxel.
        clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
      } else if (esdf_voxel->squared_distance_vox <= 1e-4) {
        // This is an invalid voxel that should be cleared.
        clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
        to_clear = true;
      }
    }
    esdf_voxel->observed = true;
  }

  __syncthreads();
  
  if (threadIdx.x == 1 && threadIdx.y == 1 && threadIdx.z == 1) {
    if (updated) {
      updated_vec[atomicAdd(updated_vec_size, 1)] = block_indices[block_idx];
    }
    if (to_clear) {
      to_clear_vec[atomicAdd(to_clear_vec_size, 1)] = block_indices[block_idx];
    }
  }
}

// From:
// https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ float atomicMinFloat(float* addr, float value) {
  float old;
  old = (value >= 0)
            ? __int_as_float(atomicMin((int*)addr, __float_as_int(value)))
            : __uint_as_float(
                  atomicMax((unsigned int*)addr, __float_as_uint(value)));

  return old;
}

/// Thread size MUST be 8x8x8, block size can be anything.
__global__ void markSitesInSliceCombinedKernel(
    int num_blocks, Index3D* block_indices,
    Index3DDeviceHashMapType<TsdfBlock> tsdf_block_hash,
    Index3DDeviceHashMapType<EsdfBlock> esdf_block_hash,
    float max_site_distance_m, float min_weight, float max_squared_distance_vox,
    int output_voxel_index, float min_height_z, float max_height_z,
    float block_size, Index3D* updated_vec, int* updated_vec_size,
    Index3D* to_clear_vec, int* to_clear_vec_size) {
  dim3 voxel_index = threadIdx;
  voxel_index.z = output_voxel_index;
  int layer_index = threadIdx.z;
  int num_layers = blockDim.z;

  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;

  __shared__ EsdfVoxel new_values[kVoxelsPerSide][kVoxelsPerSide];
  __shared__ bool observed[kVoxelsPerSide][kVoxelsPerSide];
  __shared__ float min_distance[kVoxelsPerSide][kVoxelsPerSide];

  __shared__ Index3D min_block_index, min_voxel_index, max_block_index,
      max_voxel_index;
  __shared__ Index3D esdf_block_index;
  __shared__ EsdfBlock* esdf_block;
  __shared__ bool updated, cleared;

  // Initialize these.
  if (layer_index == 0) {
    observed[voxel_index.x][voxel_index.y] = false;
    min_distance[voxel_index.x][voxel_index.y] = 2 * max_squared_distance_vox;
  }
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    updated = false;
    cleared = false;
    getBlockAndVoxelIndexFromPositionInLayer(
        block_size, Vector3f(0.0f, 0.0f, min_height_z), &min_block_index,
        &min_voxel_index);
    getBlockAndVoxelIndexFromPositionInLayer(
        block_size, Vector3f(0.0f, 0.0f, max_height_z), &max_block_index,
        &max_voxel_index);
    esdf_block_index = block_indices[blockIdx.x];
    esdf_block = nullptr;
    auto it = esdf_block_hash.find(esdf_block_index);
    if (it != esdf_block_hash.end()) {
      esdf_block = it->second;
    }
  }
  __syncthreads();

  // This shouldn't happen.
  if (esdf_block == nullptr) {
    printf(
        "No output block exists in markSitesInSliceCombinedKernel(). Shouldn't "
        "happen.\n");
    return;
  }

  // Get the correct block for this.
  Index3D tsdf_block_index = esdf_block_index;
  tsdf_block_index.z() = min_block_index.z() + layer_index;

  const TsdfBlock* tsdf_block = nullptr;
  auto it = tsdf_block_hash.find(tsdf_block_index);
  if (it != tsdf_block_hash.end()) {
    tsdf_block = it->second;
  }

  // There's also null pointers in there.
  if (tsdf_block != nullptr) {
    // Iterate over all of the voxels in this block.
    int start_index = 0;
    int end_index = kVoxelsPerSide;
    if (layer_index == 0) {
      start_index = min_voxel_index.z();
    }
    if (layer_index == num_layers - 1) {
      end_index = max_voxel_index.z();
    }
    for (int i = start_index; i < end_index; i++) {
      const TsdfVoxel* tsdf_voxel =
          &tsdf_block->voxels[voxel_index.x][voxel_index.y][i];
      // Get the correct voxel for this index.
      if (tsdf_voxel->weight >= min_weight) {
        observed[voxel_index.x][voxel_index.y] = true;
        atomicMinFloat(&min_distance[voxel_index.x][voxel_index.y],
                       tsdf_voxel->distance);
      }
    }
  }

  // sync threads across everyone trying to update this voxel
  __syncthreads();

  // Ok now only if we're layer 0 do we compare the new and old values and
  // decide what to output.
  if (layer_index == 0) {
    EsdfVoxel* esdf_voxel =
        &esdf_block->voxels[voxel_index.x][voxel_index.y][voxel_index.z];

    // Case 0: Just skip it if it's unobserved. We don't care.
    if (observed[voxel_index.x][voxel_index.y]) {
      // Determine if the new value puts us inside or in a site.
      bool is_inside = min_distance[voxel_index.x][voxel_index.y] <= 0.0f;
      bool is_site = fabsf(min_distance[voxel_index.x][voxel_index.y]) <=
                         max_site_distance_m &&
                     is_inside;

      // First handle the case where the voxel is a site.
      if (is_site) {
        if (esdf_voxel->is_site) {
          // Ok whatever. Add to the site list.
          // Its existing values are fine.
          updated = true;
        } else {
          // Wasn't a site before, is now.
          esdf_voxel->observed = true;
          esdf_voxel->is_site = true;
          clearVoxelDevice(esdf_voxel, 0.0f);
          updated = true;
        }
      } else {
        // Here we have to double-check what's going on.
        // If it was a site before, and isn't anymore, we have to clear it.
        if (esdf_voxel->is_site) {
          esdf_voxel->is_site = false;
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
          cleared = true;
        }
        // Otherwise just leave it alone unless it's brand new.
        if (!esdf_voxel->observed) {
          esdf_voxel->observed = true;
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
        } else if (esdf_voxel->is_inside != is_inside) {
          // In case the sidedness swapped, clear the voxel.
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
          cleared = true;
        } else if (esdf_voxel->squared_distance_vox <= 0.0f) {
          // This is somehow invalidly marked as a site despite the fact
          // it shouldn't be.
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
          cleared = true;
        }
      }
      // Make the sidedness match.
      esdf_voxel->is_inside = is_inside;
    }
  }

  // Now output the updated and cleared.
  __syncthreads();
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    if (updated) {
      updated_vec[atomicAdd(updated_vec_size, 1)] = esdf_block_index;
    }
    if (cleared) {
      to_clear_vec[atomicAdd(to_clear_vec_size, 1)] = esdf_block_index;
    }
  }
}

__device__ void sweepSingleBand(Index3D voxel_index, int sweep_axis,
                                float max_squared_distance_vox,
                                EsdfBlock* esdf_block) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  Index3D last_site;
  bool site_found;
  // Sweep sweep sweep.
  // First we sweep forward, then backwards.
  for (int i = 0; i < 2; i++) {
    last_site = Index3D::Zero();
    site_found = false;
    int direction = 1;
    int start_voxel = 0;
    int end_voxel = kVoxelsPerSide;
    if (i == 1) {
      direction = -1;
      start_voxel = kVoxelsPerSide - 1;
      end_voxel = -1;
    }

    for (voxel_index(sweep_axis) = start_voxel;
         voxel_index(sweep_axis) != end_voxel;
         voxel_index(sweep_axis) += direction) {
      EsdfVoxel* esdf_voxel =
          &esdf_block
               ->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
      if (!esdf_voxel->observed) {
        continue;
      }
      // If this voxel is itself a site, then mark this for future voxels.
      if (esdf_voxel->is_site) {
        last_site = voxel_index;
        site_found = true;
      } else if (!site_found) {
        // If this voxel isn't a site but we haven't found a site yet,
        // then if this voxel is valid we set it as the site.
        if (esdf_voxel->squared_distance_vox < max_squared_distance_vox) {
          site_found = true;
          last_site = esdf_voxel->parent_direction + voxel_index;
        }
      } else {
        // If we've found the site, then should just decide what to do
        // here.
        Index3D potential_direction = last_site - voxel_index;
        float potential_distance = potential_direction.squaredNorm();
        // Either it hasn't been set at all or it's closer to the site
        // than to its current value.
        if (esdf_voxel->squared_distance_vox > potential_distance) {
          esdf_voxel->parent_direction = potential_direction;
          esdf_voxel->squared_distance_vox = potential_distance;
        } else if (esdf_voxel->squared_distance_vox <
                   max_squared_distance_vox) {
          // If the current value is a better site, then set it as a site.
          last_site = esdf_voxel->parent_direction + voxel_index;
        }
      }
    }
  }
}

__device__ bool updateSingleNeighbor(const EsdfBlock* esdf_block,
                                     const Index3D& voxel_index,
                                     const Index3D& neighbor_voxel_index,
                                     int axis, int direction,
                                     float max_squared_distance_vox,
                                     EsdfBlock* neighbor_block) {
  const EsdfVoxel* esdf_voxel =
      &esdf_block->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  EsdfVoxel* neighbor_voxel =
      &neighbor_block
           ->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                   [neighbor_voxel_index.z()];
  if (!esdf_voxel->observed || !neighbor_voxel->observed ||
      neighbor_voxel->is_site ||
      esdf_voxel->squared_distance_vox >= max_squared_distance_vox) {
    return false;
  }
  // Determine if we can update this.
  Eigen::Vector3i potential_direction = esdf_voxel->parent_direction;
  potential_direction(axis) -= direction;
  float potential_distance = potential_direction.squaredNorm();
  // TODO: might be some concurrency issues here, have to be a bit careful
  // on the corners/edges.
  if (neighbor_voxel->squared_distance_vox > potential_distance) {
    neighbor_voxel->parent_direction = potential_direction;
    neighbor_voxel->squared_distance_vox = potential_distance;
    return true;
  }
  return false;
}

__device__ bool clearSingleNeighbor(const EsdfBlock* esdf_block,
                                    const Index3D& voxel_index,
                                    const Index3D& neighbor_voxel_index,
                                    int axis, int direction,
                                    float max_squared_distance_vox,
                                    EsdfBlock* neighbor_block) {
  const EsdfVoxel* esdf_voxel =
      &esdf_block->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  EsdfVoxel* neighbor_voxel =
      &neighbor_block
           ->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                   [neighbor_voxel_index.z()];

  if (esdf_voxel->squared_distance_vox < max_squared_distance_vox ||
      !esdf_voxel->observed || neighbor_voxel->is_site ||
      neighbor_voxel->squared_distance_vox >= max_squared_distance_vox) {
    return false;
  }
  // Determine if we can update this.
  Index3D parent_voxel_dir = neighbor_voxel->parent_direction;
  if ((direction > 0 && parent_voxel_dir(axis) > 0) ||
      (direction < 0 && parent_voxel_dir(axis) < 0)) {
    return false;
  }

  clearVoxelDevice(neighbor_voxel, max_squared_distance_vox);
  return true;
}

void EsdfIntegrator::markAllSitesCombined(
    const TsdfLayer& tsdf_layer, const std::vector<Index3D>& block_indices,
    EsdfLayer* esdf_layer, device_vector<Index3D>* blocks_with_sites,
    device_vector<Index3D>* cleared_blocks) {
  CHECK_NOTNULL(esdf_layer);
  CHECK_NOTNULL(blocks_with_sites);

  if (block_indices.empty()) {
    return;
  }

  // Caching.
  const float voxel_size = tsdf_layer.voxel_size();
  const float max_distance_vox = max_distance_m_ / voxel_size;
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;
  // Cache the minimum distance in metric size.
  const float max_site_distance_m = max_site_distance_vox_ * voxel_size;

  int num_blocks = block_indices.size();

  block_indices_device_ = block_indices;
  blocks_with_sites->resize(num_blocks);
  cleared_blocks->resize(num_blocks);

  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();
  if (cleared_counter_device_ == nullptr || cleared_counter_host_ == nullptr) {
    cleared_counter_device_ = make_unified<int>(MemoryType::kDevice);
    cleared_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  cleared_counter_device_.setZero();

  GPULayerView<EsdfBlock> esdf_layer_view = esdf_layer->getGpuLayerView();
  GPULayerView<TsdfBlock> tsdf_layer_view = tsdf_layer.getGpuLayerView();

  // Call the kernel.
  int dim_block = num_blocks;
  constexpr int kVoxelsPerSide = EsdfBlock::kVoxelsPerSide;
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  markAllSitesCombinedKernel<<<dim_block, dim_threads, 0, cuda_stream_>>>(
      num_blocks, block_indices_device_.data(), tsdf_layer_view.getHash().impl_,
      esdf_layer_view.getHash().impl_, max_site_distance_m, min_weight_,
      max_squared_distance_vox, blocks_with_sites->data(),
      updated_counter_device_.get(), cleared_blocks->data(),
      cleared_counter_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());

  timing::Timer pack_out_timer("esdf/integrate/mark_sites/pack_out");
  updated_counter_device_.copyTo(updated_counter_host_);
  cleared_counter_device_.copyTo(cleared_counter_host_);

  blocks_with_sites->resize(*updated_counter_host_);
  cleared_blocks->resize(*cleared_counter_host_);
  pack_out_timer.Stop();
}

void EsdfIntegrator::markSitesInSliceCombined(
    const TsdfLayer& tsdf_layer, const std::vector<Index3D>& block_indices,
    float min_z, float max_z, float output_z, EsdfLayer* esdf_layer,
    device_vector<Index3D>* updated_blocks,
    device_vector<Index3D>* cleared_blocks) {
  if (block_indices.empty()) {
    return;
  }

  // Caching.
  const float voxel_size = tsdf_layer.voxel_size();
  const float max_distance_vox = max_distance_m_ / voxel_size;
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;
  // Cache the minimum distance in metric size.
  const float max_site_distance_m = max_site_distance_vox_ * voxel_size;
  constexpr int kVoxelsPerSide = EsdfBlock::kVoxelsPerSide;

  // We are going to subsample the block_indices.
  // We need to figure out all the output blocks, which will be a subset
  // of the input blocks.

  // Ok first figure out how many layers we could have.
  Index3D min_block_index;
  Index3D min_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(tsdf_layer.block_size(),
                                           Vector3f(0.0f, 0.0f, min_z),
                                           &min_block_index, &min_voxel_index);
  const int min_block_index_z = min_block_index.z();
  const int min_voxel_index_z = min_voxel_index.z();
  Index3D max_block_index;
  Index3D max_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(tsdf_layer.block_size(),
                                           Vector3f(0.0f, 0.0f, max_z),
                                           &max_block_index, &max_voxel_index);
  const int max_block_index_z = max_block_index.z();
  const int max_voxel_index_z = max_voxel_index.z();

  // And figure out what the index of the output voxel is.
  // std::pair<Index3D, Index3D> output_block_and_voxel_index
  Index3D output_block_index;
  Index3D output_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(
      tsdf_layer.block_size(), Vector3f(0.0f, 0.0f, output_z),
      &output_block_index, &output_voxel_index);
  const int output_block_index_z = output_block_index.z();
  const int output_voxel_index_z = output_voxel_index.z();

  // There is always at least 1 layer.
  int num_vertical_layers = max_block_index_z - min_block_index_z + 1;
  CHECK_GE(max_block_index_z, min_block_index_z);

  // Next get a list of all the valid input blocks.
  Index3DSet output_block_set;
  for (const Index3D& block_index : block_indices) {
    if (block_index.z() >= min_block_index_z &&
        block_index.z() <= max_block_index_z) {
      output_block_set.insert(
          Index3D(block_index.x(), block_index.y(), output_block_index_z));
    }
  }

  // Resize everything to the final size.
  size_t num_blocks = output_block_set.size();
  block_indices_host_.resize(num_blocks);
  updated_indices_device_.resize(num_blocks);
  to_clear_indices_device_.resize(num_blocks);

  if (num_blocks == 0) {
    return;
  }

  // Reset the counters.
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();
  if (cleared_counter_device_ == nullptr || cleared_counter_host_ == nullptr) {
    cleared_counter_device_ = make_unified<int>(MemoryType::kDevice);
    cleared_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  cleared_counter_device_.setZero();

  // Convert to vector and allocate output blocks.
  size_t i = 0;
  for (const Index3D& block_index : output_block_set) {
    // This is for the output block, which we allocate along the way.
    block_indices_host_[i] = block_index;
    esdf_layer->allocateBlockAtIndex(block_index);

    // Go through all the relevant input pointers:
    Index3D input_block_index = block_index;
    i++;
  }
  block_indices_device_ = block_indices_host_;

  // Get the GPU hash of both the TSDF and the ESDF.
  GPULayerView<EsdfBlock> esdf_layer_view = esdf_layer->getGpuLayerView();
  GPULayerView<TsdfBlock> tsdf_layer_view = tsdf_layer.getGpuLayerView();

  // Figure out the size of the kernel.
  int dim_block = num_blocks;
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, num_vertical_layers);
  // Call the kernel!
  markSitesInSliceCombinedKernel<<<dim_block, dim_threads, 0, cuda_stream_>>>(
      num_blocks, block_indices_device_.data(), tsdf_layer_view.getHash().impl_,
      esdf_layer_view.getHash().impl_, max_site_distance_m, min_weight_,
      max_squared_distance_vox, output_voxel_index.z(), min_z, max_z,
      tsdf_layer.block_size(), updated_blocks->data(),
      updated_counter_device_.get(), cleared_blocks->data(),
      cleared_counter_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());
}

__host__ __device__ void getDirectionAndVoxelIndicesFromThread(
    dim3 thread_index, Index3D* block_direction, Index3D* voxel_index,
    Index3D* neighbor_voxel_index, int* axis, int* direction) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  *block_direction = Index3D::Zero();
  // Thread index z is used for the neighbor number. x & y are actual voxel
  // indices.
  (*block_direction)(thread_index.z / 2) = thread_index.z % 2 ? -1 : 1;
  // This is to to make it easier to pick voxel directions.
  *axis = thread_index.z / 2;
  *direction = thread_index.z % 2 ? -1 : 1;

  // Fill in the axes.
  if (*axis == 0) {
    *voxel_index << 0, thread_index.x, thread_index.y;
  } else if (*axis == 1) {
    *voxel_index << thread_index.x, 0, thread_index.y;
  } else if (*axis == 2) {
    *voxel_index << thread_index.x, thread_index.y, 0;
  }
  *neighbor_voxel_index = *voxel_index;
  // If we're looking backwards...
  if (*direction < 0) {
    (*voxel_index)(*axis) = 0;
    (*neighbor_voxel_index)(*axis) = kVoxelsPerSide - 1;
  } else {
    (*voxel_index)(*axis) = kVoxelsPerSide - 1;
    (*neighbor_voxel_index)(*axis) = 0;
  }
}

// Thread size MUST be 8x8x6, 8x8 being the side of the cube, and 6 being the
// number of neighbors considered per block. Block size can be whatever.
__global__ void updateNeighborBandsCombinedKernel(
    int i, int num_blocks, Index3DDeviceHashMapType<EsdfBlock> block_hash,
    float max_squared_distance_vox, Index3D* block_indices,
    Index3D* output_vector, int* updated_size) {
  // For every block in the initial list, look up its neighbors.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  constexpr int kNumNeighbors = 6;

  // Luckily the direction is the same for all processed blocks by this thread.
  Index3D block_direction, voxel_index, neighbor_voxel_index;
  int axis, direction;
  getDirectionAndVoxelIndicesFromThread(threadIdx, &block_direction,
                                        &voxel_index, &neighbor_voxel_index,
                                        &axis, &direction);

  __shared__ bool block_updated;
  // Allow block size to be whatever.
  __shared__ EsdfBlock* block_ptr;
  EsdfBlock* neighbor_block_ptr = nullptr;
  for (int block_idx = blockIdx.x; block_idx < num_blocks;
       block_idx += gridDim.x) {
    __syncthreads();
    // Get the current block for this... block.
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
      block_ptr = nullptr;
      auto it = block_hash.find(block_indices[block_idx]);
      if (it != block_hash.end()) {
        block_ptr = it->second;
      }
      block_updated = false;
    }
    __syncthreads();
    // This block doesn't exist. Who knows why. This shouldn't happen.
    if (block_ptr == nullptr) {
      continue;
    }

    dim3 specific_thread = threadIdx;
    specific_thread.z = i;
    Index3D block_direction, voxel_index, neighbor_voxel_index;
    int axis, direction;
    getDirectionAndVoxelIndicesFromThread(specific_thread, &block_direction,
                                          &voxel_index, &neighbor_voxel_index,
                                          &axis, &direction);

    // Get the neighbor block for this thread.
    neighbor_block_ptr = nullptr;
    auto it = block_hash.find(block_indices[block_idx] + block_direction);
    if (it != block_hash.end()) {
      neighbor_block_ptr = it->second;
    }
    // Our neighbor doesn't exist. This is fine and normal. Happens to
    // everyone.
    if (neighbor_block_ptr == nullptr) {
      continue;
    }

    bool updated = updateSingleNeighbor(
        block_ptr, voxel_index, neighbor_voxel_index, axis, direction,
        max_squared_distance_vox, neighbor_block_ptr);
    // No bother with atomics.
    if (updated) {
      block_updated = updated;
    }

    __syncthreads();
    if ((threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) &&
        block_updated) {
      //*any_updated = true;
      output_vector[atomicAdd(updated_size, 1)] =
          block_indices[block_idx] + block_direction;
    }
  }
}

template <int kBlockThreads, int kItemsPerThread>
__global__ void sortUniqueKernel(Index3D* indices, int num_indices,
                                 int* num_output_indices) {
  typedef uint64_t IndexHashValue;
  typedef int OriginalIndex;

  typedef hipcub::BlockRadixSort<uint64_t, kBlockThreads, kItemsPerThread,
                              OriginalIndex>
      BlockRadixSortT;
  typedef hipcub::BlockDiscontinuity<IndexHashValue, kBlockThreads>
      BlockDiscontinuityT;
  typedef hipcub::BlockScan<OriginalIndex, kBlockThreads> BlockScanT;

  // Allocate type-safe, repurposable shared memory for collectives
  __shared__ union {
    typename BlockRadixSortT::TempStorage sort;
    typename BlockDiscontinuityT::TempStorage discontinuity;
    typename BlockScanT::TempStorage scan;
  } temp_storage;

  // First we create a values list which is actually the indicies.
  // Obtain this block's segment of consecutive keys (blocked across threads)
  uint64_t thread_keys[kItemsPerThread];
  Index3D thread_values[kItemsPerThread];
  int thread_inds[kItemsPerThread];
  int head_flags[kItemsPerThread];
  int head_indices[kItemsPerThread];
  int thread_offset = threadIdx.x * kItemsPerThread;

  // Fill in the keys from the values.
  // I guess we can just do a for loop. kItemsPerThread should be fairly small.
  Index3DHash index_hash;
  for (int i = 0; i < kItemsPerThread; i++) {
    if (thread_offset + i >= num_indices) {
      // We just pack the key with a large value.
      thread_values[i] = Index3D::Zero();
      thread_keys[i] = SIZE_MAX;
      thread_inds[i] = -1;
    } else {
      thread_values[i] = indices[thread_offset + i];
      thread_keys[i] = index_hash(thread_values[i]);
      thread_inds[i] = thread_offset + i;
    }
  }

  // We then sort the values.
  __syncthreads();
  // Collectively sort the keys
  BlockRadixSortT(temp_storage.sort).Sort(thread_keys, thread_inds);
  __syncthreads();
  // We remove duplicates by find when the discontinuities happen.
  BlockDiscontinuityT(temp_storage.discontinuity)
      .FlagHeads(head_flags, thread_keys, hipcub::Inequality());
  __syncthreads();
  // Get the indices that'll be assigned to the new unique values.
  BlockScanT(temp_storage.scan)
      .InclusiveSum<kItemsPerThread>(head_flags, head_indices);
  __syncthreads();

  // Cool now write only 1 instance of the unique entries to the output.
  for (int i = 0; i < kItemsPerThread; i++) {
    if (thread_offset + i < num_indices) {
      if (head_flags[i] == 1) {
        // Get the proper value out. Cache this for in-place ops next step.
        thread_values[i] = indices[thread_inds[i]];
        atomicMax(num_output_indices, head_indices[i]);
      }
    }
  }
  __syncthreads();

  // Have to do this twice since we do this in-place. Now actually replace
  // the values.
  for (int i = 0; i < kItemsPerThread; i++) {
    if (thread_offset + i < num_indices) {
      if (head_flags[i] == 1) {
        // Get the proper value out.
        indices[head_indices[i] - 1] = thread_values[i];
      }
    }
  }
}

void EsdfIntegrator::sortAndTakeUniqueIndices(
    device_vector<Index3D>* block_indices) {
  if (block_indices->size() == 0) {
    return;
  }
  // Together this should be >> the number of indices
  constexpr int kNumThreads = 128;
  constexpr int kNumItemsPerThread = 4;
  if (block_indices->size() >= kNumThreads * kNumItemsPerThread) {
    LOG(INFO) << "Vector too big to sort. Falling back to thrust.";
    // sort vertices to bring duplicates together
    thrust::sort(thrust::device, block_indices->begin(), block_indices->end(),
                 VectorCompare<Index3D>());

    // Find unique vertices and erase redundancies. The iterator will point to
    // the new last index.
    auto iterator = thrust::unique(thrust::device, block_indices->begin(),
                                   block_indices->end());

    // Figure out the new size.
    size_t new_size = iterator - block_indices->begin();
    block_indices->resize(new_size);
    return;
  }
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();

  sortUniqueKernel<kNumThreads, kNumItemsPerThread>
      <<<1, kNumThreads, 0, cuda_stream_>>>(block_indices->data(),
                                            block_indices->size(),
                                            updated_counter_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));

  updated_counter_device_.copyTo(updated_counter_host_);
  block_indices->resize(*updated_counter_host_*2);
}

// Combined functions.
void EsdfIntegrator::updateNeighborBandsCombined(
    device_vector<Index3D>* block_indices, EsdfLayer* esdf_layer,
    float max_squared_distance_vox,
    device_vector<Index3D>* updated_block_indices) {
  if (block_indices->empty()) {
    return;
  }
  timing::Timer sweep_timer("esdf/integrate/compute/neighbor_bands");

  // This function just copies neighbors across block boundaries.
  constexpr int kNumNeighbors = 6;
  constexpr int kUpdatedBlockMultiple = kNumNeighbors;
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;

  updated_block_indices->resize(block_indices->size() * kUpdatedBlockMultiple);
  updated_block_indices->setZero();

  // Create an output variable.
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();

  timing::Timer gpu_view("esdf/integrate/compute/neighbor_bands/gpu_view");
  GPULayerView<EsdfBlock> gpu_layer_view = esdf_layer->getGpuLayerView();
  gpu_view.Stop();

  // Call the kernel.
  int dim_block = block_indices->size();
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, 1);
  for (int i = 0; i < kNumNeighbors; i++) {
    updateNeighborBandsCombinedKernel<<<dim_block, dim_threads, 0,
                                        cuda_stream_>>>(
        i, block_indices->size(), gpu_layer_view.getHash().impl_,
        max_squared_distance_vox, block_indices->data(),
        updated_block_indices->data(), updated_counter_device_.get());
  }
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));

  updated_counter_device_.copyTo(updated_counter_host_);
  updated_block_indices->resize(*updated_counter_host_);

  if (*updated_counter_host_ == 0) {
    return;
  }

  timing::Timer copy_out_timer(
      "esdf/integrate/compute/neighbor_bands/copy_out");
  sortAndTakeUniqueIndices(updated_block_indices);
}

/// Thread size MUST be 8x8xN (where N is a number of blocks up to ???), block
/// size can be anything.
__global__ void sweepBlockBandCombinedKernel(
    int num_blocks, Index3DDeviceHashMapType<EsdfBlock> block_hash,
    float max_squared_distance_vox, Index3D* block_indices) {
  // We go one axis at a time, syncing threads in between.
  dim3 thread_index = threadIdx;
  thread_index.z = 0;

  __shared__ EsdfBlock* esdf_block;

  for (int block_idx = blockIdx.x * blockDim.z + threadIdx.z;
       block_idx < num_blocks; block_idx += gridDim.x * blockDim.z) {
    // For simplicity we have to have the same number of blocks in the CUDA
    // kernel call as we have actual blocks.
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      esdf_block = nullptr;
      auto it = block_hash.find(block_indices[block_idx]);
      if (it != block_hash.end()) {
        esdf_block = it->second;
      }
    }
    __syncthreads();
    // This block doesn't exist. Who knows why. This shouldn't happen.
    if (esdf_block == nullptr) {
      continue;
    }
    Index3D voxel_index(0, thread_index.x, thread_index.y);

    // X axis done.
    sweepSingleBand(voxel_index, 0, max_squared_distance_vox, esdf_block);
    __syncthreads();

    // Y axis done.
    voxel_index << thread_index.x, 0, thread_index.y;
    sweepSingleBand(voxel_index, 1, max_squared_distance_vox, esdf_block);
    __syncthreads();

    // Z axis done.
    voxel_index << thread_index.x, thread_index.y, 0;
    sweepSingleBand(voxel_index, 2, max_squared_distance_vox, esdf_block);
    __syncthreads();
  }
}

void EsdfIntegrator::sweepBlockBandCombined(
    device_vector<Index3D>* block_indices, EsdfLayer* esdf_layer,
    float max_squared_distance_vox) {
  if (block_indices->empty()) {
    return;
  }
  timing::Timer sweep_timer("esdf/integrate/compute/sweep");

  // Caching.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const int num_blocks = block_indices->size();

  GPULayerView<EsdfBlock> gpu_layer_view = esdf_layer->getGpuLayerView();

  // Call the kernel.
  // We do 2-dimensional sweeps in this kernel. Each thread does 3 sweeps.
  // We do 1 blocks at a time because it's faster.
  constexpr int kNumBlocksPerCudaBlock = 1;
  int dim_block = std::max(
      static_cast<int>(
          std::ceil(num_blocks / static_cast<float>(kNumBlocksPerCudaBlock))),
      1);
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kNumBlocksPerCudaBlock);
  sweepBlockBandCombinedKernel<<<dim_block, dim_threads, 0, cuda_stream_>>>(
      block_indices->size(), gpu_layer_view.getHash().impl_,
      max_squared_distance_vox, block_indices->data());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());
}

void EsdfIntegrator::computeEsdfCombined(
    const device_vector<Index3D>& blocks_with_sites, EsdfLayer* esdf_layer) {
  CHECK_NOTNULL(esdf_layer);

  if (blocks_with_sites.size() == 0) {
    return;
  }
  // Cache everything.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const float voxel_size = esdf_layer->block_size() / kVoxelsPerSide;
  const float max_distance_vox = max_distance_m_ / voxel_size;
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;

  // First we go over all of the blocks with sites.
  // We compute all the proximal sites inside the block first.
  block_indices_device_ = blocks_with_sites;
  sweepBlockBandCombined(&block_indices_device_, esdf_layer,
                         max_squared_distance_vox);

  int i = 0;
  while (!block_indices_device_.empty()) {
    updateNeighborBandsCombined(&block_indices_device_, esdf_layer,
                                max_squared_distance_vox,
                                &updated_indices_device_);
    sweepBlockBandCombined(&updated_indices_device_, esdf_layer,
                           max_squared_distance_vox);

    timing::Timer swap_timer("esdf/integrate/compute/swap");
    std::swap(block_indices_device_, updated_indices_device_);
    swap_timer.Stop();
  }
}

__device__ void getBlockAndVoxelIndexFromOffset(const Index3D& block_index,
                                                const Index3D& voxel_index,
                                                const Index3D& voxel_offset,
                                                Index3D* neighbor_block_index,
                                                Index3D* neighbor_voxel_index) {
  // For each axis we have to get the mod and div to get the block index and
  // voxel index.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;

  for (int i = 0; i < 3; i++) {
    (*neighbor_block_index)(i) =
        block_index(i) + voxel_offset(i) / kVoxelsPerSide;
    (*neighbor_voxel_index)(i) =
        voxel_index(i) + voxel_offset(i) % kVoxelsPerSide;
    if ((*neighbor_voxel_index)(i) >= kVoxelsPerSide) {
      (*neighbor_voxel_index)(i) -= kVoxelsPerSide;
      (*neighbor_block_index)(i)++;
    } else if ((*neighbor_voxel_index)(i) < 0) {
      (*neighbor_voxel_index)(i) += kVoxelsPerSide;
      (*neighbor_block_index)(i)--;
    }
  }
}

__global__ void clearAllInvalidKernel(
    Index3D* block_indices, Index3DDeviceHashMapType<EsdfBlock> block_hash,
    float max_squared_distance_vox, Index3D* output_vector, int* updated_size) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  __shared__ int block_updated;
  // Allow block size to be whatever.
  __shared__ EsdfBlock* block_ptr;
  // Get the current block for this... block.
  __shared__ Index3D block_index;
  Index3D voxel_index = Index3D(threadIdx.x, threadIdx.y, threadIdx.z);
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    block_ptr = nullptr;
    block_index = block_indices[blockIdx.x];
    auto it = block_hash.find(block_index);
    if (it != block_hash.end()) {
      block_ptr = it->second;
    }
    block_updated = false;
  }
  __syncthreads();
  // This block doesn't exist. Who knows why. This shouldn't happen.
  if (block_ptr == nullptr) {
    return;
  }

  // Now for our specific voxel we should look up its parent and see if it's
  // still there.
  EsdfVoxel* esdf_voxel =
      &block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];

  if (esdf_voxel->observed && !esdf_voxel->is_site &&
      esdf_voxel->parent_direction != Index3D::Zero()) {
    Index3D neighbor_block_index, neighbor_voxel_index;
    getBlockAndVoxelIndexFromOffset(
        block_index, voxel_index, esdf_voxel->parent_direction,
        &neighbor_block_index, &neighbor_voxel_index);

    EsdfVoxel* neighbor_voxel = nullptr;
    if (neighbor_block_index == block_index) {
      neighbor_voxel =
          &block_ptr->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                            [neighbor_voxel_index.z()];
    } else {
      // Get the neighboring block.
      auto it = block_hash.find(neighbor_block_index);
      if (it != block_hash.end()) {
        neighbor_voxel =
            &it->second
                 ->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                         [neighbor_voxel_index.z()];
      }
    }
    if (neighbor_voxel != nullptr && !neighbor_voxel->is_site) {
      // Clear this voxel.
      esdf_voxel->parent_direction.setZero();
      esdf_voxel->squared_distance_vox = max_squared_distance_vox;
      block_updated = true;
    }
  }
  __syncthreads();
  if ((threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) &&
      block_updated) {
    output_vector[atomicAdd(updated_size, 1)] = block_index;
  }
}

void EsdfIntegrator::clearAllInvalid(
    const std::vector<Index3D>& blocks_to_clear, EsdfLayer* esdf_layer,
    device_vector<Index3D>* updated_blocks) {
  if (blocks_to_clear.size() == 0) {
    return;
  }

  // TODO: start out just getting all the blocks in the whole map.
  // Then replace with blocks within a radius of the cleared blocks.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const float max_distance_vox = max_distance_m_ / esdf_layer->voxel_size();
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;

  timing::Timer get_blocks_timer("esdf/integrate/clear/get_blocks");

  temp_indices_host_ = getBlocksWithinRadiusOfAABB(
      esdf_layer->getAllBlockIndices(), esdf_layer->block_size(),
      getAABBOfBlocks(esdf_layer->block_size(), blocks_to_clear),
      max_distance_m_);
  get_blocks_timer.Stop();
  temp_indices_device_ = temp_indices_host_;

  // Get the hash map of the whole ESDF map.
  GPULayerView<EsdfBlock> gpu_layer_view = esdf_layer->getGpuLayerView();

  // Create an output variable.
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();

  // Make sure we have enough space if EVERYTHING had to be cleared.
  updated_blocks->resize(temp_indices_device_.size());

  // Call a kernel.
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  if (temp_indices_device_.size() > 0) {
    clearAllInvalidKernel<<<temp_indices_device_.size(), dim_threads, 0,
                            cuda_stream_>>>(
        temp_indices_device_.data(), gpu_layer_view.getHash().impl_,
        max_squared_distance_vox, updated_blocks->data(),
        updated_counter_device_.get());
    checkCudaErrors(hipStreamSynchronize(cuda_stream_));
    checkCudaErrors(hipPeekAtLastError());

    // Pack out the updated blocks.
    updated_counter_device_.copyTo(updated_counter_host_);
    updated_blocks->resize(*updated_counter_host_);
  } else {
    updated_blocks->resize(0);
  }
}

}  // namespace nvblox
